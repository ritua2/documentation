#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
using namespace std;

int main(){

 int count, err;
 hipGetDeviceCount(&count);
 cout<<"device count = "<<count<<"\n";
 if(err = hipSetDevice(count-1)){
    cout<<"hipSetDevice error, "<<err<<"\n";
 }
 return 0;
}

