#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>

int main(){

 int count, err;
 hipGetDeviceCount(&count);
 printf("device count = %d\n", count); 
 if(err = hipSetDevice(count-1)){
    printf("hipSetDevice error, %d\n", err);
 }
 return 0;
}

